#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2014/2015, March 2 - April 3
// ###
// ### Project Name: Poisson Image Editing
// ### Group members: Gaurav Krishna Joshi, Saion Chatterjee
// ### 
// ### Supervisor: Thomas Moellenhoff
// ###
// ###
// ### Note for user: Please use same image format and same dimension for all the 3 input images- source, mask and target
// ###


#include "aux.h"
#include <iostream>
#include "math.h"
using namespace std;

//  uncomment to use the camera
//#define CAMERA

//  uncomment any one of the following to run the corresponding version- CPU, GAUSS, SOR or SHARED
//  WARNING: Do not uncomment multiple versions below. Only one of them must be uncommented.
//#define CPU
//#define GAUSS
#define SOR
//#define SHARED

//  uncomment any one of the following to run with corresponding guiding gradient- source or mixed or no guiding gradient
//  WARNING: Do not uncomment multiple versions below. Only one of them must be uncommented.
//#define SOURCE_GRADIENT
#define MIXED_GRADIENT
//#define NO_GRADIENT

//  Iteration value- can also be passed from the command line
#define ITERATIONS 7000

//  please do not comment the below parameters
#define THETA 0.9

#define INSIDE_MASK           0
#define BOUNDRY               1
#define OUTSIDE               2

#define CORNER_PIXEL_0_0      3
#define CORNER_PIXEL_0_H      4 
#define CORNER_PIXEL_W_0      5
#define CORNER_PIXEL_W_H      6 

#define EDGE_PIXEL_RIGHT      7 
#define EDGE_PIXEL_LEFT       8
#define EDGE_PIXEL_UP         9
#define EDGE_PIXEL_DOWN       10 


// Source Image Masking Kernel. 
__global__ void SourceImageMasking(float *Imgsrc, float *Mask, float *ImgOut_Srcmask, int nc, int w ,int h)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  for (int channel=0;channel < nc ;channel++)
	{
		int id=x + w*y + w*h*channel;

		//Normalizing mask id for any coloured pixels if left out during manual masking
		if(Mask[id]<0.5)
			Mask[id]=0;
		else
			Mask[id]=1;

        //Creating the image with only the masked portion
		ImgOut_Srcmask[id]=Imgsrc[id]*Mask[id];
	}   
}



// Extracting Boundry Pixel using the mask. 
__global__ void ExtractingBoundryPixels(float *Mask, int *BoundryPixelsArray,int nc, int w ,int h)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  for (int channel=0;channel < nc ;channel++)
	{
       if(x<w && y<h)
            {
			int id=x + w*y +w*h*channel;

			if(x==0 && y==0 && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = CORNER_PIXEL_0_0;  
			  }
			else if(x==0 && y==(h-1) && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = CORNER_PIXEL_0_H;  
			  }
			else if(x==(w-1) && y==0 && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = CORNER_PIXEL_W_0;  
			  }
			else if(x==(w-1) && y==(h-1) && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = CORNER_PIXEL_W_H;  
			  }
			else if(x==0 && y<(h-1) && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = EDGE_PIXEL_LEFT;  
			  }
			else if(x==(w-1) && y<(h-1) && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = EDGE_PIXEL_RIGHT;  
			  }
			else if(x<(w-1) && y==0 && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = EDGE_PIXEL_DOWN;  
			  }
			else if(x<(w-1) && y==(h-1) && Mask[id]==1)
			  {
			      BoundryPixelsArray[id] = EDGE_PIXEL_UP;  
			  }
                else
                  {  
			  int id_r=x+1 + w*y +w*h*channel;
			  int id_l=x-1 + w*y +w*h*channel;
			  int id_u=x + w*(y+1) +w*h*channel;
			  int id_d=x + w*(y-1) +w*h*channel;

			  if(Mask[id]==1 && Mask[id_r]==1 && Mask[id_l]==1 && Mask[id_u]==1 && Mask[id_d]==1)
				 {
				    BoundryPixelsArray[id]=INSIDE_MASK;   // Totally Inside 
				 }
			  else if((Mask[id]==1) && (Mask[id_r]==0 || Mask[id_l]==0 || Mask[id_u]==0 || Mask[id_d]==0))
				 {
				    BoundryPixelsArray[id]=BOUNDRY;  //Boundry
				 }
   	                  else
	                         {
	                            BoundryPixelsArray[id]=OUTSIDE;  //Totally outside
	                         }
                  }
            }
      }
}



//Calculate boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY the bound box variables for the selected region.
void calculate_boundBoxMinMax(int w, int h, int nc, int *BoundaryPixelsArray, int *boundBoxMinX, int *boundBoxMinY, int *boundBoxMaxX, int *boundBoxMaxY)
{
	//Initializing variables
	*boundBoxMinX=99999;
	*boundBoxMinY=99999;
	*boundBoxMaxX=0;
	*boundBoxMaxY=0;

	//Calculating boundBoxMinX and boundBoxMinY
	for(int c=0;c<nc;c++)
	{
		for(int x=0;x<=w-1;x++)
		{
			for(int y=0;y<=h-1;y++)
			{
				int idx = x + w*y +w*h*c;

				if(BoundaryPixelsArray[idx]==BOUNDRY)
				{
					  if(x < *boundBoxMinX)
					  {
					      *boundBoxMinX = x;
					  }
					  if(x > *boundBoxMaxX)
					  {
					       *boundBoxMaxX = x;
					  }
					  if(y < *boundBoxMinY)
					  {
					      *boundBoxMinY = y;
					  }
					  if(y > *boundBoxMaxY)
					  {
					       *boundBoxMaxY = y;
					  }          
		                }  
		        }
	       }
     }
}

//Test Kernel to verify image boundaries- This is only for testing purpose and so no need to call this.
__global__ void BoundryTest(float *Imgin, int *BoundryPixelsArray ,float *ImgOut, int nc, int w ,int h)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  for (int channel=0;channel < nc ;channel++)
    {
		int id=x + w*y +w*h*channel;

		if(BoundryPixelsArray[id]==BOUNDRY)
		{
			ImgOut[id]=1;
		}
		else
		{
			ImgOut[id]=Imgin[id];
		}
    }   
}

//Kernel to show NormalCloning. Simple copy paste of the selected region without any blending technique.
__global__ void SourceMaskImageMergeinTargetImage(float *Imgsrc, float *Imgtarget, float *ImgOut, int nc, int w ,int h)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  for (int channel=0;channel < nc ;channel++)
    {
		int id=x + w*y +w*h*channel;

        if(Imgsrc[id]==INSIDE_MASK)
           {
               ImgOut[id]=Imgtarget[id];
           }
        else
           {
               ImgOut[id]=Imgsrc[id];
           }
    }   
}

// Initializing the Output image.
__global__ void initialize(float *t, float *u, int *boundary, float *s, int w , int h, int nc)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    //Initialization for Dirichlet boundary conditions
   for(int c=0; c<nc; c++)
   {
		int idx = x + w*y +w*h*c;

		if(boundary[idx]==BOUNDRY || boundary[idx]==OUTSIDE) 
			u[idx]=t[idx];        // u is the final output image and t is the input target image
		else
			u[idx]=0;
	}
}

// Evaluating the Guiding Gradient of the output image's selected region
__global__ void evaluate_gradient(float *v_l, float *v_r, float *v_d, float *v_u, int *boundary, float *s, float *t, int target_nc, int w, int h)
{
   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;
    
   if(x<w && y<h)//Boundary of entire pixel grid
	   {
		  for(int c=0; c<target_nc; c++)
		  {
			  int idx = x + w*y +w*h*c;
			  int idx_nextX = x+1 + w*y +w*h*c;
			  int idx_prevX = x-1 + w*y + w*h*c;
			  int idx_nextY = x + w*(y+1) +w*h*c;
			  int idx_prevY = x + w*(y-1) +w*h*c; 

#ifdef NO_GRADIENT
			  //For No guiding gradient approach
			  v_r[idx] = 0;
			  v_l[idx] = 0;
			  v_u[idx] = 0;
			  v_d[idx] = 0;
#endif

#ifdef SOURCE_GRADIENT
	                 //For Source gradient approach

                         switch (boundary[idx]) //Required for handling for corner pixels of the image grid
		            {
					case INSIDE_MASK:  
						    v_r[idx] = s[idx]-s[idx_nextX];
						    v_l[idx] = s[idx]-s[idx_prevX];
						    v_u[idx] = s[idx]-s[idx_prevY];
						    v_d[idx] = s[idx]-s[idx_nextY];  
					break;
					case CORNER_PIXEL_0_0:
						    v_l[idx] = 0;
							v_u[idx] = 0;
						    v_r[idx] = s[idx]-s[idx_nextX];
						    v_d[idx] = s[idx]-s[idx_nextY];  

					break;      
					case CORNER_PIXEL_0_H:
							v_l[idx] = 0;
							v_d[idx] = 0;
						    v_r[idx] = s[idx]-s[idx_nextX];
						    v_u[idx] = s[idx]-s[idx_prevY];

					break;       
					case CORNER_PIXEL_W_0:
							v_r[idx] = 0;
							v_u[idx] = 0;
						    v_l[idx] = s[idx]-s[idx_prevX];
						    v_d[idx] = s[idx]-s[idx_nextY];  

					break;
					case CORNER_PIXEL_W_H: 
							v_r[idx] = 0;
							v_d[idx] = 0;
						    v_l[idx] = s[idx]-s[idx_prevX];
						    v_u[idx] = s[idx]-s[idx_prevY];

					break;

					case  EDGE_PIXEL_RIGHT:
							v_r[idx] = 0;
						    v_l[idx] = s[idx]-s[idx_prevX];
						    v_u[idx] = s[idx]-s[idx_prevY];
						    v_d[idx] = s[idx]-s[idx_nextY];  

					break;
					case  EDGE_PIXEL_LEFT:
							v_l[idx] = 0;
						    v_r[idx] = s[idx]-s[idx_nextX];
						    v_u[idx] = s[idx]-s[idx_prevY];
						    v_d[idx] = s[idx]-s[idx_nextY];  

					break;    
					case  EDGE_PIXEL_UP:
							v_d[idx] = 0;
						    v_r[idx] = s[idx]-s[idx_nextX];
						    v_l[idx] = s[idx]-s[idx_prevX];
						    v_u[idx] = s[idx]-s[idx_prevY];

					break;
					case  EDGE_PIXEL_DOWN:
							v_u[idx] = 0;
						    v_r[idx] = s[idx]-s[idx_nextX];
						    v_l[idx] = s[idx]-s[idx_prevX];
						    v_d[idx] = s[idx]-s[idx_nextY];  

				break; 
	    	   }
#endif  //Endif for Source gradient

#ifdef MIXED_GRADIENT
       //For Mixed gradient approach    
		
       //Declaring the source s and target t variables to store the relative gradient of the neighbouring pixels in 4-directions
		   float s_diff_r, t_diff_r;
		   float s_diff_l, t_diff_l;
		   float s_diff_d, t_diff_d;
		   float s_diff_u, t_diff_u;

       switch (boundary[idx]) //Required for handling for corner pixels of the image grid
	            {
					case INSIDE_MASK:  

							s_diff_r = s[idx]-s[idx_nextX]; 
						    s_diff_l = s[idx]-s[idx_prevX];
						    s_diff_u = s[idx]-s[idx_prevY];
						    s_diff_d = s[idx]-s[idx_nextY];  

							t_diff_r = t[idx]-t[idx_nextX]; 
						    t_diff_l = t[idx]-t[idx_prevX];
						    t_diff_u = t[idx]-t[idx_prevY];
						    t_diff_d = t[idx]-t[idx_nextY];  

					break;

					case CORNER_PIXEL_0_0:
							s_diff_r = s[idx]-s[idx_nextX]; 
						    s_diff_l = 0;
						    s_diff_u = 0;
						    s_diff_d = s[idx]-s[idx_nextY];  

							t_diff_r = t[idx]-t[idx_nextX]; 
						    t_diff_l = 0;
						    t_diff_u = 0;
						    t_diff_d = t[idx]-t[idx_nextY];  

					break;      
					case CORNER_PIXEL_0_H:
							s_diff_r = s[idx]-s[idx_nextX]; 
						    s_diff_l = 0;
						    s_diff_u = s[idx]-s[idx_prevY];
						    s_diff_d = 0;

							t_diff_r = t[idx]-t[idx_nextX]; 
						    t_diff_l = 0;
						    t_diff_u = t[idx]-t[idx_prevY];
						    t_diff_d = 0;

					break;       
					case CORNER_PIXEL_W_0:
							s_diff_r = 0;
						    s_diff_l = s[idx]-s[idx_prevX];
						    s_diff_u = 0;
						    s_diff_d = s[idx]-s[idx_nextY];  

							t_diff_r = 0;
						    t_diff_l = t[idx]-t[idx_prevX];
						    t_diff_u = 0;
						    t_diff_d = t[idx]-t[idx_nextY];  

					break;
					case CORNER_PIXEL_W_H: 
							s_diff_r = 0;
						    s_diff_l = s[idx]-s[idx_prevX];
						    s_diff_u = s[idx]-s[idx_prevY];
						    s_diff_d = 0;

							t_diff_r = 0;
						    t_diff_l = t[idx]-t[idx_prevX];
						    t_diff_u = t[idx]-t[idx_prevY];
						    t_diff_d = 0;

					break;

					case  EDGE_PIXEL_RIGHT:
						s_diff_r = 0;
						    s_diff_l = s[idx]-s[idx_prevX];
						    s_diff_u = s[idx]-s[idx_prevY];
						    s_diff_d = s[idx]-s[idx_nextY];  

							t_diff_r = 0;
						    t_diff_l = t[idx]-t[idx_prevX];
						    t_diff_u = t[idx]-t[idx_prevY];
						    t_diff_d = t[idx]-t[idx_nextY];  

					break;
					case  EDGE_PIXEL_LEFT:
							s_diff_r = s[idx]-s[idx_nextX]; 
						    s_diff_l = 0;
						    s_diff_u = s[idx]-s[idx_prevY];
						    s_diff_d = s[idx]-s[idx_nextY];  

							t_diff_r = t[idx]-t[idx_nextX]; 
						    t_diff_l = 0;
						    t_diff_u = t[idx]-t[idx_prevY];
						    t_diff_d = t[idx]-t[idx_nextY];  

					break;    
					case  EDGE_PIXEL_UP:
							s_diff_r = s[idx]-s[idx_nextX]; 
						    s_diff_l = s[idx]-s[idx_prevX];
						    s_diff_u = s[idx]-s[idx_prevY];
						    s_diff_d = 0;

							t_diff_r = t[idx]-t[idx_nextX]; 
						    t_diff_l = t[idx]-t[idx_prevX];
						    t_diff_u = t[idx]-t[idx_prevY];
						    t_diff_d = 0;

					break;
					case  EDGE_PIXEL_DOWN:
							s_diff_r = s[idx]-s[idx_nextX]; 
						    s_diff_l = s[idx]-s[idx_prevX];
						    s_diff_u = 0;
						    s_diff_d = s[idx]-s[idx_nextY];  

							t_diff_r = t[idx]-t[idx_nextX]; 
						    t_diff_l = t[idx]-t[idx_prevX];
						    t_diff_u = 0;
						    t_diff_d = t[idx]-t[idx_nextY];  

					break; 
	    	   }

//After appropriate evaluation of relative gradient of s and t, now comparing and assigning greater abs value for mixed gradient approach

						  if(abs(s_diff_r) > abs(t_diff_r))
						  	v_r[idx] = s_diff_r;
						  else
							v_r[idx] = t_diff_r;

						  if(abs(s_diff_d) > abs(t_diff_d))
							 v_d[idx] = s_diff_d;
						  else
							 v_d[idx] = t_diff_d;

						  if(abs(s_diff_l) > abs(t_diff_l))
							 v_l[idx] = s_diff_l;
						  else
							 v_l[idx] = t_diff_l;

						  if(abs(s_diff_u) > abs(t_diff_u))
							 v_u[idx] = s_diff_u;
						  else
							 v_u[idx] = t_diff_u;

#endif // endif for mixed gradient
		        
			 }
	   }

}


// Gauss Seidel implementation- GPU
__global__ void poisson_gauss_seidel(int *boundary, float *u, int w , int h,int target_nc,int boundBoxMinX, int boundBoxMinY, int boundBoxMaxX, int boundBoxMaxY, float *v_l, float *v_r, float *v_d, float *v_u)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x + boundBoxMinX;
    int y = threadIdx.y + blockDim.y * blockIdx.y + boundBoxMinY;
    
	if(x<=boundBoxMaxX && y<=boundBoxMaxY)//Boundary of selected bound box region
  	 {
  	for(int c=0; c<target_nc; c++)
		{
			int idx = x + w*y +w*h*c;
			int idx_nextX = x+1 + w*y +w*h*c;
			int idx_prevX = x-1 + w*y + w*h*c;
			int idx_nextY = x + w*(y+1) +w*h*c;
			int idx_prevY = x + w*(y-1) +w*h*c;  
          
        switch(boundary[idx]) 
           {
                case INSIDE_MASK  :
	   	      u[idx] = 0.25*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
        		break; 
		case CORNER_PIXEL_0_0:
			u[idx] = 0.5*(u[idx_nextX]+u[idx_nextY]+v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break;      
		case CORNER_PIXEL_0_H:
			u[idx] = 0.5*(u[idx_nextX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break;        
		case CORNER_PIXEL_W_0:
			u[idx] = 0.5*(u[idx_prevX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break; 
		case CORNER_PIXEL_W_H: 
			u[idx] = 0.5*(u[idx_prevX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break; 
		case  EDGE_PIXEL_RIGHT:
			u[idx] = 0.33*(u[idx_prevX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break; 
		case  EDGE_PIXEL_LEFT:
			u[idx] = 0.33*(u[idx_nextX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break; 
		case  EDGE_PIXEL_UP:
			u[idx] = 0.33*(u[idx_nextX]+u[idx_prevX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break; 
		case  EDGE_PIXEL_DOWN:
			u[idx] = 0.33*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
			break; 
          }
	}
  }
}

//Copying Image to solution- needed for SOR implementation
__global__ void CopyImageoutinSolution(float *Imgout, float *solution, int nc, int w ,int h)
{

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
  	for(int c=0; c<nc; c++)
		{
			int idx = x + w*y +w*h*c;
			solution[idx]=Imgout[idx];
       }
}

//SOR redblack implementation- GPU
__global__ void poisson_sor_redblack(int *boundary, float *u, float *solution, int w , int h, int target_nc, int boundBoxMinX, int boundBoxMinY, int boundBoxMaxX, int boundBoxMaxY, float *v_l, float *v_r, float *v_d, float *v_u, int param)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x + boundBoxMinX;
    int y = threadIdx.y + blockDim.y * blockIdx.y + boundBoxMinY;

    float result=0; //Using local variables for storing intermediate values to reduce array lookup for optimization purpose
    
    float theta = THETA;
    
if(x<=boundBoxMaxX && y<=boundBoxMaxY)//Boundary of selected bound box region
   {
  	for(int c=0; c<target_nc; c++)
	      {
		int idx = x + w*y +w*h*c;
		int idx_nextX = x+1 + w*y +w*h*c;
		int idx_prevX = x-1 + w*y + w*h*c;
		int idx_nextY = x + w*(y+1) +w*h*c;
		int idx_prevY = x + w*(y-1) +w*h*c; 

	    switch(boundary[idx])
	       {
                case INSIDE_MASK  :

		if((x+y)%2 == 0 && param == 0 /*Red step*/)
			{ 
				result = 0.25*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
				solution[idx] =  result + theta*(result - solution[idx]) ;
			        // storing the result in the current pixel Optimization
			        u[idx]=result;
			}
		if((x+y)%2 == 1 && param == 1 /*Black step*/)
			{   
				result = 0.25*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
				solution[idx] =  result + theta*(result - solution[idx]) ;    
			        // storing the result in the current pixel Optimization
			        u[idx]=result;
			}

			break; 

		case CORNER_PIXEL_0_0:

			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.5*(u[idx_nextX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result = 0.5*(u[idx_nextX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;    
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}

   		        break;   

		case CORNER_PIXEL_0_H:

			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.5*(u[idx_nextX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result = 0.5*(u[idx_nextX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;    
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}
		    break;      

		case CORNER_PIXEL_W_0:

			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.5*(u[idx_prevX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result = 0.5*(u[idx_prevX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;    
				       // storing the result in the current pixel Optimization
				       u[idx]=result;  
				}
		    break;

		case CORNER_PIXEL_W_H: 
			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.5*(u[idx_prevX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
				       // storing the result in the current pixel Optimization
				       u[idx]=result;  
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result = 0.5*(u[idx_prevX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;    
				       // storing the result in the current pixel Optimization
				       u[idx]=result;  
				}
		    break;

		case  EDGE_PIXEL_RIGHT:
			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.33*(u[idx_prevX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
		       		       // storing the result in the current pixel Optimization
		       		       u[idx]=result; 	
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result = 0.33*(u[idx_prevX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;    
		       		       // storing the result in the current pixel Optimization
		       		       u[idx]=result; 							
				}

		    break;

		case  EDGE_PIXEL_LEFT:
			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.33*(u[idx_nextX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result= 0.33*(u[idx_nextX]+u[idx_nextY]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;    
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}
		    break;      

		case  EDGE_PIXEL_UP:
			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.33*(u[idx_nextX]+u[idx_prevX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
				        // storing the result in the current pixel Optimization
				        u[idx]=result; 
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result = 0.33*(u[idx_nextX]+u[idx_prevX]+u[idx_prevY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;   
				        // storing the result in the current pixel Optimization
				        u[idx]=result;  
				}

		    break;

		case  EDGE_PIXEL_DOWN:
			if((x+y)%2 == 0 && param == 0 /*Red step*/)
				{ 
					result = 0.33*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ;
				       // storing the result in the current pixel Optimization
				       u[idx]=result; 
				}
			if((x+y)%2 == 1 && param == 1 /*Black step*/)
				{   
					result = 0.33*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+ v_r[idx]+v_l[idx]+v_u[idx]+v_d[idx]);
					solution[idx] =  result + theta*(result - solution[idx]) ; 
				       // storing the result in the current pixel Optimization
				       u[idx]=result;    
				}
		    break; 
        	   }
  
	      }
  	 }
}

//SOR redblack implementation using Shared memory- GPU
__global__ void poisson_shared_sor_redblack(int *boundary, float *u, float *solution, int w , int h, int target_nc, int boundBoxMinX, int boundBoxMinY, int boundBoxMaxX, int boundBoxMaxY, float *v_l, float *v_r, float *v_d, float *v_u, int param)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = blockIdx.z; //Using z-dimension for channels instead of for loop

    float theta = THETA; //Used in SOR
    float result = 0;    //Using local variable to minimize global memory access for optimization

    extern __shared__ float share[];

    if(x<w && y<h)
	   {
   	  //Code to populate the shared memory
	  int share_height = (2 + blockDim.y); //Square shared memory block
	  int share_width = (2 + blockDim.x); //Square shared memory block
	  
	  int bw = blockDim.x; //block_width
	  int bx = blockDim.x * blockIdx.x; //block index for starting point of each block in x-dimension
	  int by = blockDim.y * blockIdx.y; //block index for starting point of each block in y-dimension
 

	  int ind = threadIdx.x + bw * threadIdx.y; //Thread index inside each block

	  int  sidx_y, sidx_x;  //Shared memory index variables

	 //Populating the shared memory
	 	for (int i = ind; i < share_height * share_width; i += (blockDim.x * blockDim.y))
		{
		  sidx_y = by - 1 + i / share_width;
		  sidx_x = bx - 1 + i % share_width;

		  sidx_x = max(min(sidx_x, w-1), 0);  //Clamping shared memory
		  sidx_y = max(min(sidx_y, h-1), 0);

		  share[i] = u[sidx_x + w * sidx_y + w * h * z];
		}

	 __syncthreads();

   
  //Code for SOR red-black poisson blending using the populated Shared memory

  //indexes for shared memory
	//int idx = (threadIdx.x+1) + ((threadIdx.y+1) * share_width); //idx is not used below so commenting this, it is only for understanding.
	int idx_nextX = (threadIdx.x+1+1) + ((threadIdx.y+1) * share_width);
	int idx_prevX = (threadIdx.x) + ((threadIdx.y+1) * share_width);
	int idx_nextY = (threadIdx.x+1) + ((threadIdx.y+1+1) * share_width);
	int idx_prevY = (threadIdx.x+1) + ((threadIdx.y) * share_width);

        int gid = x + w*y + w*h*z; //Global index
      

    switch(boundary[gid])
       {
       case INSIDE_MASK:

    		if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.25*(share[idx_nextX]+share[idx_prevX]+share[idx_nextY]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

         	u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.25*(share[idx_nextX]+share[idx_prevX]+share[idx_nextY]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;

      case CORNER_PIXEL_0_0:
	 
    		if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.5*(share[idx_nextX]+share[idx_nextY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.5*(share[idx_nextX]+share[idx_nextY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;

      case CORNER_PIXEL_0_H:

    		if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.5*(share[idx_nextX]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.5*(share[idx_nextX]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;       

	case CORNER_PIXEL_W_0:

    		if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.5*(share[idx_prevX]+share[idx_nextY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.5*(share[idx_prevX]+share[idx_nextY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;

	case CORNER_PIXEL_W_H: 

    		if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.5*(share[idx_prevX]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.5*(share[idx_prevX]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;

	case  EDGE_PIXEL_RIGHT:

    		if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.33*(share[idx_prevX]+share[idx_nextY]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.33*(share[idx_prevX]+share[idx_nextY]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;

	case  EDGE_PIXEL_LEFT:

	    	if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.33*(share[idx_nextX]+share[idx_nextY]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.33*(share[idx_nextX]+share[idx_nextY]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;      

	case  EDGE_PIXEL_UP:

	    	if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.33*(share[idx_nextX]+share[idx_prevX]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

	        u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.33*(share[idx_nextX]+share[idx_prevX]+share[idx_prevY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break;

	case  EDGE_PIXEL_DOWN:

	    	if((x+y)%2 == 0 && param == 0 ) //Red step
		{ 
		result = 0.33*(share[idx_nextX]+share[idx_prevX]+share[idx_nextY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

	        u[gid] = result;
		}

		if((x+y)%2 == 1 && param == 1) //Black step
		{   
		result = 0.33*(share[idx_nextX]+share[idx_prevX]+share[idx_nextY]+ v_r[gid]+v_l[gid]+v_u[gid]+v_d[gid]);

	    	solution[gid] =  result + theta*(result - solution[gid]) ;

		u[gid] = result;
		}

		break; 
	       }
    }
}


//CPU Poisson simple
void poisson_cpu(int *f, float *s, float *u, float *t, int w , int h,int c,int iterations )
{
	int idx, idx_nextX,idx_nextY,idx_prevX,idx_prevY;
	for(int i=0; i<iterations; i++)
		{
	  for(int x=0; x<=w-1; x++)
	  {
		 for(int y=0; y<=h-1; y++)
			{
				idx = x + w*y +w*h*c;
				idx_nextX = x+1 + w*y +w*h*c;
				idx_prevX = x-1 + w*y + w*h*c;
				idx_nextY = x + w*(y+1) +w*h*c;
				idx_prevY = x + w*(y-1) +w*h*c;            


		//if we take guiding gradient as 0    
			/*	 if(f[idx]==0 && x<w-1 && y<h-1 && x>0 && y>0) 
					   u[idx] = 0.25*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+u[idx_prevY]);
                        */

		//if we take guiding gradient as source image gradient
				if(f[idx]==0 && x<w-1 && y<h-1 && x>0 && y>0) 
					u[idx] = 0.25*(u[idx_nextX]+u[idx_prevX]+u[idx_nextY]+u[idx_prevY]+4*s[idx]-s[idx_nextX]-s[idx_nextY]-s[idx_prevX]-s[idx_prevY]);

			}
	 }
		}
}



int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  
    CUDA_CHECK;

    int iterations=ITERATIONS;

    // input from command prompt :  source_image ,target_image and mask
    string source_image = "";
    string mask = "";

    bool ret;

    ret = getParam("it", iterations, argc, argv);
    if (!ret) cerr << "No Iteration Count passed" << endl;
    cout<<" Iteration Count   : "<<iterations<<endl;

    ret = getParam("s", source_image, argc, argv);
    if (!ret) cerr << "ERROR: no source_image specified" << endl;
    cout<<" source_image   : "<<source_image<<endl;

    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << " gray: " << gray << endl;

#ifdef CAMERA
    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;

  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mTargetImage;
    camera >> mTargetImage;
#else
    string target_image = "";
    ret = getParam("t", target_image, argc, argv);
    if (!ret) cerr << "ERROR: no target_image specified" << endl;
    cout<<" target_image   : "<<target_image<<endl;
    // Load the input source_image using opencv 
    cv::Mat mTargetImage = cv::imread(target_image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
#endif

    // check for Target Image
    if (mTargetImage.data == NULL) { cerr << "ERROR: Could not load target image "<< endl; return 1; }

    ret = getParam("m", mask, argc, argv);
    if (!ret) cerr << "ERROR: no mask specified" << endl;
    cout<<" Mask name   : "<<mask <<endl;

    if (argc <= 1) { cout << "Usage: " << argv[0] << " -s <source_image> -t <target_image>  -m <mask>  [-it <iterations>] [-repeats <repeats>] [-gray]" << endl; return 1; }
   
    // Load the input source_image using opencv 
    cv::Mat mSourceImage = cv::imread(source_image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    if (mSourceImage.data == NULL) { cerr << "ERROR: Could not load source image " << source_image << endl; return 1; }

    // Load the input source_image using opencv 
    cv::Mat mmask = cv::imread(mask.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    if (mmask.data == NULL) { cerr << "ERROR: Could not load mask image " << mask << endl; return 1; }


    // convert to float representation (opencv loads image values as single bytes by default)
    mSourceImage.convertTo(mSourceImage,CV_32F);
    mTargetImage.convertTo(mTargetImage,CV_32F);
    mmask.convertTo(mmask,CV_32F);

    // convert range of each channel to [0,1] (opencv default is [0,255])
    mSourceImage /= 255.f;
    mTargetImage /= 255.f;
    mmask /= 255.f;

    // get source image dimensions
    int source_w = mSourceImage.cols;         // width
    int source_h = mSourceImage.rows;         // height
    int source_nc = mSourceImage.channels();  // number of channels
    cout <<endl<<" Source image   : " << source_w << " x " << source_h << " x " <<source_nc<<endl;

    // get target image dimensions
    int target_w = mTargetImage.cols;         // width
    int target_h = mTargetImage.rows;         // height
    int target_nc = mTargetImage.channels();  // number of channels
    cout <<endl<<" target image  : " << target_w << " x " << target_h << " x " <<target_nc<<endl;

    // get source image dimensions
    int mask_w = mmask.cols;         // width
    int mask_h = mmask.rows;         // height
    int mask_nc = mmask.channels();  // number of channels
    cout <<endl<<" mask          : " << mask_w << " x " << mask_h << " x " <<mask_nc<<endl;


    // Output Images
    cv::Mat mOutSourceImgMasked(target_h,target_w,mTargetImage.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOutShiftSourceImgMasked(target_h,target_w,mTargetImage.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOut(target_h,target_w,mTargetImage.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOutBoundryCheck(target_h,target_w,mTargetImage.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOutNormalClone(target_h,target_w,mTargetImage.type());  // mOut will have the same number of channels as the input image, nc layers


    // allocate raw input source image array
    float *srcimgIn  = new float[(size_t)source_w*source_h*source_nc];
    // allocate raw input mask image array
    float *maskIn  = new float[(size_t)mask_w*mask_h*mask_nc];

    convert_mat_to_layered (srcimgIn, mSourceImage);
    convert_mat_to_layered (maskIn, mmask);


    // Display Source Image and Mask 
    convert_layered_to_mat(mSourceImage, srcimgIn);
    showImage("SourceImage", mSourceImage, 200, 200); 
    convert_layered_to_mat(mmask, maskIn);
    showImage("mask", mmask, 200, 200); 


    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut_Srcmask = new float[(size_t)target_w*target_h*mOutSourceImgMasked.channels()];
    float *imgOutBoundryCheck = new float[(size_t)target_w*target_h*mOut.channels()];
    int *boundryPixelArray = new int[(size_t)target_w*target_h*mOut.channels()];
    float *targetimgIn  = new float[(size_t)target_w*target_h*target_nc];
    float *imgOutNormalClone = new float[(size_t)target_w*target_h*mOutSourceImgMasked.channels()];
    float *imgOut = new float[(size_t)target_w*target_h*mOut.channels()];

    ///////////////////////// Start of GPU IMPLEMENTATION /////////////////////////////
    float *d_srcimgIn;
    float *d_mask;
    float *d_targetimgIn;
    float *d_imgOut_Srcmask;
    float *d_imgOutNormalClone;
    float *d_imgOut;
    float *d_imgOutBoundryCheck;
    int *d_boundryPixelArray;
    float *d_solution;
    float *d_v_l, *d_v_r, *d_v_d, *d_v_u; //Declaring relative gradient variables

    // Allocating memory
    hipMalloc( &d_srcimgIn, source_w*source_h*source_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_mask, mask_w*mask_h*mask_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_imgOut_Srcmask, target_w*target_h*target_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_solution, target_w*target_h*target_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_imgOutBoundryCheck, target_w*target_h*target_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_boundryPixelArray, target_w*target_h*target_nc * sizeof(int) );
    CUDA_CHECK;
    hipMalloc( &d_targetimgIn, target_w*target_h*target_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_imgOutNormalClone, target_w*target_h*target_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_imgOut, target_w*target_h*target_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_v_l, source_w*source_h*source_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_v_r, source_w*source_h*source_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_v_d, source_w*source_h*source_nc * sizeof(float) );
    CUDA_CHECK;
    hipMalloc( &d_v_u, source_w*source_h*source_nc * sizeof(float) );
    CUDA_CHECK;

    // copying memory to GPU Source Image and Mask
    hipMemcpy( d_srcimgIn, srcimgIn, source_w*source_h*source_nc* sizeof(float), hipMemcpyHostToDevice ); 
    CUDA_CHECK;
    hipMemcpy( d_mask, maskIn, mask_w*mask_h*mask_nc* sizeof(float), hipMemcpyHostToDevice ); 
    CUDA_CHECK;

    // Processing : Launching Kernels
    
    dim3 block (32,4,1); 
    dim3 grid = dim3( (target_w+ block.x - 1) / block.x , (target_h + block.y - 1 ) / block.y ,1 );

#ifdef SHARED
    //dim variable declarations for shared memory
    dim3 block_shared (32,4,1); 
    dim3 grid_shared = dim3((target_w+ block.x - 1) / block.x , (target_h + block.y - 1) / block.y , 3 );

    size_t shared = (2 + block_shared.x) * (2 + block_shared.y) *sizeof(float);
#endif

    // Extracting desired portion of the image using Mask
    SourceImageMasking <<< grid , block >>> (d_srcimgIn ,d_mask, d_imgOut_Srcmask , source_nc, source_w, source_h);  

  
    hipMemcpy( imgOut_Srcmask, d_imgOut_Srcmask, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
    CUDA_CHECK;
    convert_layered_to_mat(mOutSourceImgMasked, imgOut_Srcmask);
    showImage("Source_Image_Merged_with_Mask", mOutSourceImgMasked, 200, 200); 


    // Extracting Boundry Pixels
    ExtractingBoundryPixels <<< grid , block >>> (d_mask ,d_boundryPixelArray , source_nc, source_w, source_h);   
    
    //Calculating now BoundboxMinMaxXY using cpu call
    hipDeviceSynchronize();
    CUDA_CHECK;

    hipMemcpy(boundryPixelArray , d_boundryPixelArray, target_nc*target_w*target_h * sizeof(float), hipMemcpyDeviceToHost ); 
    CUDA_CHECK;
    
    int boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY; //Declaring CPU boundary variables

    calculate_boundBoxMinMax(target_w, target_h, target_nc, boundryPixelArray, &boundBoxMinX, &boundBoxMinY, &boundBoxMaxX, &boundBoxMaxY);

    hipDeviceSynchronize();
    CUDA_CHECK;

    hipMemcpy(boundryPixelArray, d_boundryPixelArray, source_nc*source_w*source_h * sizeof(int), hipMemcpyDeviceToHost ); 
    CUDA_CHECK;


#ifndef CPU //if not CPU version then only the below variables will be declared
    int selected_w, selected_h;     //Declaring selected region variables

    dim3 grid_selected; //declaring for launching kernel for only selected region- the boundBox approach
#endif

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(300) < 0)
    {
	    camera >> mTargetImage;
	    // convert to float representation (opencv loads image values as single bytes by default)
	    mTargetImage.convertTo(mTargetImage,CV_32F);
	    // convert range of each channel to [0,1] (opencv default is [0,255])
	    mTargetImage /= 255.f;
#endif
        convert_mat_to_layered (targetimgIn, mTargetImage);

        // Copying from Host to Device the Target Image inside while loop to allow for new dynamic target images when used with webcam
	    hipMemcpy( d_targetimgIn, targetimgIn, target_w*target_h*target_nc* sizeof(float), hipMemcpyHostToDevice ); 
	    CUDA_CHECK;

	// Pasting desired portion of the source image to Target Image- Normal Cloning without any blending technique
	    SourceMaskImageMergeinTargetImage <<< grid , block >>> (d_imgOut_Srcmask ,d_targetimgIn, d_imgOutNormalClone , target_nc, target_w, target_h); 
	    hipMemcpy( imgOutNormalClone, d_imgOutNormalClone, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
	    CUDA_CHECK;
        //End of Normal Clone

   	// Boundry Test- Only for Test purpose- Hence Commented
	/* BoundryTest <<< grid , block >>> (d_imgOut ,d_boundryPixelArray, d_imgOutBoundryCheck , target_nc, target_w, target_h); 
	   hipMemcpy( imgOutBoundryCheck, d_imgOutBoundryCheck, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
           CUDA_CHECK;
	   convert_layered_to_mat(mOutBoundryCheck, imgOutBoundryCheck);
	   showImage("FinalImage_BoundryCheck", mOutBoundryCheck, 200, 200); 
        */

        //For initializing the Output image with Dirichlet Boundary conditions
	    initialize <<< grid , block >>> (d_targetimgIn, d_imgOut, d_boundryPixelArray, d_imgOut_Srcmask, target_w , target_h, target_nc);    
       CUDA_CHECK;

#ifndef CPU  //if not for CPU version then do the following steps. 
       //Call Kernel to calculate the guiding gradient. This is one-time activity and so done outside the iteration loop for optimization.
 	    evaluate_gradient <<< grid, block >>> (d_v_l, d_v_r, d_v_d, d_v_u, d_boundryPixelArray, d_imgOut_Srcmask, d_targetimgIn, target_nc, target_w, target_h); 
        CUDA_CHECK;

#ifdef SHARED
	  CopyImageoutinSolution <<< grid , block >>> (d_imgOut, d_solution, target_nc, target_w, target_h);
      CUDA_CHECK;
#endif


#ifdef SOR
       CopyImageoutinSolution <<< grid , block >>> (d_imgOut, d_solution, target_nc, target_w, target_h);
       CUDA_CHECK;
#endif

	   selected_w = boundBoxMaxX - boundBoxMinX + 1;
	   selected_h = boundBoxMaxY - boundBoxMinY + 1;

	   grid_selected = dim3((selected_w+ block.x - 1) / block.x , (selected_h + block.y - 1 ) / block.y ,1 ); //Defining new grid for only selected region- the boundBox

	   float time_gpu;
	   Timer timer_GPU; 
           timer_GPU.start();


	   for(int i=0; i<iterations; i++)
	   {   

#ifdef GAUSS
	//Gauss Siedel
   	       poisson_gauss_seidel  <<< grid_selected , block >>> (d_boundryPixelArray , d_imgOut, target_w , target_h, target_nc, boundBoxMinX, boundBoxMinY,boundBoxMaxX, boundBoxMaxY, d_v_l, d_v_r, d_v_d, d_v_u);
               CUDA_CHECK;
#endif

#ifdef SOR
  
     	//SOR Red black scheme
		poisson_sor_redblack  <<< grid_selected , block >>> (d_boundryPixelArray , d_imgOut, d_solution, target_w , target_h, target_nc, boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY, d_v_l, d_v_r, d_v_d, d_v_u, 0); //RED

		poisson_sor_redblack  <<< grid_selected , block >>> (d_boundryPixelArray , d_imgOut, d_solution, target_w , target_h, target_nc, boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY, d_v_l, d_v_r, d_v_d, d_v_u, 1); //BLACK
                
               CUDA_CHECK;
#endif

#ifdef SHARED
//SOR Red black scheme with shared memory
            	poisson_shared_sor_redblack  <<< grid_shared , block_shared, shared >>> (d_boundryPixelArray , d_imgOut, d_solution, target_w , target_h, target_nc, boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY, d_v_l, d_v_r, d_v_d, d_v_u, 0); //RED
 
            	poisson_shared_sor_redblack  <<< grid_shared , block_shared, shared >>> (d_boundryPixelArray , d_imgOut, d_solution, target_w , target_h, target_nc, boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY, d_v_l, d_v_r, d_v_d, d_v_u, 1); //BLACK

                CUDA_CHECK;
#endif

	   }

       timer_GPU.end();  
       time_gpu = timer_GPU.get();  // elapsed time in seconds
       cout << "Time GPU : " << time_gpu*1000 << " ms" << endl;
#ifdef SHARED
	//copy result back to host (CPU) memory
         hipMemcpy( imgOut, d_solution, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 	
	     CUDA_CHECK;
#endif

#ifdef SOR 
	//copy result back to host (CPU) memory
	    hipMemcpy( imgOut, d_solution, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
	    CUDA_CHECK;
#endif

#ifdef GAUSS 
	    hipMemcpy( imgOut, d_imgOut, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
	    CUDA_CHECK;
#endif


#else  //Else part for CPU Poisson implementation
  

	hipDeviceSynchronize();
	CUDA_CHECK;

	hipMemcpy( boundryPixelArray, d_boundryPixelArray, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
	CUDA_CHECK;

	hipMemcpy( imgOut_Srcmask, d_imgOut_Srcmask, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
	CUDA_CHECK;

	hipMemcpy( targetimgIn, d_targetimgIn, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
	CUDA_CHECK;

	hipMemcpy( imgOut, d_imgOut, source_nc*source_w*source_h * sizeof(float), hipMemcpyDeviceToHost ); 
	CUDA_CHECK;

	hipDeviceSynchronize();
	CUDA_CHECK;

       float time_cpu;
       Timer timer_CPU; 
       timer_CPU.start();

		for(int c=0; c<target_nc; c++)
          {
		    poisson_cpu(boundryPixelArray , imgOut_Srcmask , imgOut, targetimgIn, target_w , target_h,c,iterations);

          }
       timer_CPU.end();  
       time_cpu = timer_CPU.get();  // elapsed time in seconds
       cout << "time CPU : " << time_cpu*1000 << " ms" << endl;

#endif //Endif for CPU Poisson implementation

	convert_layered_to_mat(mTargetImage, targetimgIn);
	showImage("targetImage", mTargetImage, 200, 200); 

	convert_layered_to_mat(mOutNormalClone, imgOutNormalClone);
	showImage("NormalClone", mOutNormalClone, 200, 200); 

	convert_layered_to_mat(mOut, imgOut);
	showImage("FinalImage", mOut, 800, 200); 

#ifdef CAMERA
	    // end of camera loop
 }

#else
    cv::imwrite("mask_input.jpg",mmask*255.f);
    // save input and result
    cv::imwrite("Sourceimage_input.jpg",mSourceImage*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("targetimage_input.jpg",mTargetImage*255.f);
    cv::imwrite("Source_Image_Merged_with_Mask.jpg",mOutSourceImgMasked*255.f);
    cv::imwrite("FinalImage.jpg",mOut*255.f);

    // wait for key inputs
    cv::waitKey(1000000);
#endif

    // free allocated arrays and GPU arrays
    hipFree(d_targetimgIn);
    CUDA_CHECK;
    delete[] targetimgIn;

    hipFree(d_imgOut);
    CUDA_CHECK;
    delete[] imgOut;

    hipFree(d_imgOutNormalClone);
    CUDA_CHECK;
    delete[] imgOutNormalClone;

    hipFree(d_srcimgIn);
	CUDA_CHECK;
    delete[] srcimgIn;

    hipFree(d_mask);
    CUDA_CHECK;
    delete[] maskIn;

    hipFree(d_imgOut_Srcmask);
    CUDA_CHECK;
    delete[] imgOut_Srcmask;

    hipFree(d_imgOutBoundryCheck);
    CUDA_CHECK;
    delete[] imgOutBoundryCheck;

    hipFree(d_boundryPixelArray);
    CUDA_CHECK;
    delete[] boundryPixelArray;

    hipFree(d_solution);
    CUDA_CHECK;

    hipFree(d_v_l);
    CUDA_CHECK;
    hipFree(d_v_r);
    CUDA_CHECK;
    hipFree(d_v_d);
    CUDA_CHECK;
    hipFree(d_v_u);
    CUDA_CHECK;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



